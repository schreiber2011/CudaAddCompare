#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <chrono>
#include <iostream>
#include <ctime>

using std::cout; using std::endl;
using std::chrono::duration_cast;
using std::chrono::milliseconds;
using std::chrono::seconds;
using std::chrono::system_clock;

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size, unsigned int loop);

__global__ void addKernel(int* c, const int* a, const int* b, unsigned int loop)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
    for (size_t j = 0; j < loop; j++)
    {
        c[i]++;
    }
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    unsigned int loop = 100 * 1000;

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize, loop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size, unsigned int loop)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    // Choose ... END

    // Allocate GPU buffers for three vectors (two input, one output)    .
    auto millisec_since_epoch = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();

    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    auto dif_millisec_since_epoch = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count() - millisec_since_epoch;
    cout << "Malloc time in milliseconds: " << dif_millisec_since_epoch << endl;
    // End of allocation

    // Copy input vectors from host memory to GPU buffers.
    millisec_since_epoch = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    dif_millisec_since_epoch = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count() - millisec_since_epoch;
    cout << "Copy time in milliseconds: " << dif_millisec_since_epoch << endl;
    // Copy ... END

        /******************************************************************
                                       GPU sum
            Launch a kernel on the GPU with one thread for each element.

        ******************************************************************/
    for (size_t i = 1; i < 200; i = i * 10)
    {
        millisec_since_epoch = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
        cout << "Loop count: " << loop * i << endl;
        addKernel << <1, size >> > (dev_c, dev_a, dev_b, loop * i);
        hipDeviceSynchronize();
        dif_millisec_since_epoch = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count() - millisec_since_epoch;
        cout << "Add time in milliseconds iterations " << loop * i << ": " << dif_millisec_since_epoch << endl;
    }

    // TODO: Divide task into smaller chunks
    /*
    for (size_t i = 1; i < 200; i = i * 10)
    {
        millisec_since_epoch = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
        cout << loop * i << endl;
        addKernel << <1, size >> > (dev_c, dev_a, dev_b, loop * i);
        hipDeviceSynchronize();
        dif_millisec_since_epoch = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count() - millisec_since_epoch;
        cout << "Add time in milliseconds iterations " << loop * i << ": " << dif_millisec_since_epoch << endl;
    }
    */
    // TODO (in progress) ... END


    /**********************************************
                          CPU sum
    **********************************************/
    int cpu_mutiplier = 10;
    for (size_t i = cpu_mutiplier; i < 200 * cpu_mutiplier; i = i * 10)
    {
        millisec_since_epoch = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
        cout << "Loop count: " << loop * i << ", size of array: " << size << endl;
        for (size_t k = 0; k < size; k++)
        {
            c[k] = a[k] + b[k];
            for (size_t j = 0; j < (loop * i); j++)
            {
                c[k]++;
            }
        }
        dif_millisec_since_epoch = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count() - millisec_since_epoch;
        cout << "Add (CPU) time in milliseconds iterations " << loop * i << ": " << dif_millisec_since_epoch << endl;
    }
    // END ... cpu sum

// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
